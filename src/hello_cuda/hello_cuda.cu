#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void test(void)
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    int m = threadIdx.x;
    int n = threadIdx.y;
    printf("hello CUDA %d, %d, %d, %d!\n", i, j, m, n);
}

int main()
{
    dim3 gridShape = dim3(10, 10); // blocks / grid
    dim3 blockShape = dim3(4, 4); // threads / block
    test<<<gridShape, blockShape>>>();
    hipDeviceSynchronize();
    return 0;
}