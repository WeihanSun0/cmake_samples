#include "hip/hip_runtime.h"
#include "cuda_static_lib.cuh"

#define CHECK(res) { if(res != hipSuccess){printf("Error ：%s:%d , ", __FILE__,__LINE__);   \
                    printf("code : %d , reason : %s \n", res,hipGetErrorString(res));exit(-1);}}

__global__ void foo()
{int i = blockIdx.x;
    int j = blockIdx.y;
    int m = threadIdx.x;
    int n = threadIdx.y;
    printf("hello CUDA static %d, %d, %d, %d!\n", i, j, m, n);
}

void useCUDA_static()
{
    dim3 gridShape = dim3(10, 10); // blocks / grid
    dim3 blockShape = dim3(4, 4); // threads / block
    foo<<<gridShape, blockShape>>>();
    CHECK(hipDeviceSynchronize());
}